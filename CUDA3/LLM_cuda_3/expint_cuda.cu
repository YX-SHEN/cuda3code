#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

// GPU device 端算法完全等价CPU
__device__ double expint_double_dev(int n, double x, int maxIterations) {
    const double eulerConstant = 0.5772156649015329;
    double epsilon = 1.E-30, bigDouble = 1.7976931348623157e+308;
    int i, ii, nm1 = n - 1;
    double a, b, c, d, del, fact, h, psi, ans = 0.0;
    if (n <= 0 || x < 0 || (x == 0.0 && (n == 0 || n == 1))) return -1.0;
    if (n == 0) return exp(-x) / x;
    if (x > 1.0) {
        b = x + n; c = bigDouble; d = 1.0 / b; h = d;
        for (i = 1; i <= maxIterations; i++) {
            a = -i * (nm1 + i); b += 2.0; d = 1.0 / (a * d + b);
            c = b + a / c; del = c * d; h *= del;
            if (fabs(del - 1.0) <= epsilon) return h * exp(-x);
        }
        return h * exp(-x);
    } else {
        ans = (nm1 != 0 ? 1.0 / nm1 : -log(x) - eulerConstant); fact = 1.0;
        for (i = 1; i <= maxIterations; i++) {
            fact *= -x / i;
            if (i != nm1) del = -fact / (i - nm1);
            else {
                psi = -eulerConstant;
                for (ii = 1; ii <= nm1; ii++) psi += 1.0 / ii;
                del = fact * (-log(x) + psi);
            }
            ans += del;
            if (fabs(del) < fabs(ans) * epsilon) return ans;
        }
        return ans;
    }
}
__device__ float expint_float_dev(int n, float x, int maxIterations) {
    const float eulerConstant = 0.5772156649015329f;
    float epsilon = 1.E-30f, bigfloat = 3.402823466e+38f;
    int i, ii, nm1 = n - 1;
    float a, b, c, d, del, fact, h, psi, ans = 0.0f;
    if (n <= 0 || x < 0 || (x == 0.0f && (n == 0 || n == 1))) return -1.0f;
    if (n == 0) return expf(-x) / x;
    if (x > 1.0f) {
        b = x + n; c = bigfloat; d = 1.0f / b; h = d;
        for (i = 1; i <= maxIterations; i++) {
            a = -i * (nm1 + i); b += 2.0f; d = 1.0f / (a * d + b);
            c = b + a / c; del = c * d; h *= del;
            if (fabsf(del - 1.0f) <= epsilon) return h * expf(-x);
        }
        return h * expf(-x);
    } else {
        ans = (nm1 != 0 ? 1.0f / nm1 : -logf(x) - eulerConstant); fact = 1.0f;
        for (i = 1; i <= maxIterations; i++) {
            fact *= -x / i;
            if (i != nm1) del = -fact / (i - nm1);
            else {
                psi = -eulerConstant;
                for (ii = 1; ii <= nm1; ii++) psi += 1.0f / ii;
                del = fact * (-logf(x) + psi);
            }
            ans += del;
            if (fabsf(del) < fabsf(ans) * epsilon) return ans;
        }
        return ans;
    }
}

// GPU kernel，每个线程计算 (i, j) 一个结果
__global__ void expint_kernel(int n, int m, double a, double b, int maxIterations,
                              float *resultsFloat, double *resultsDouble) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int total = n * m;
    if (tid < total) {
        int i = tid / m;      // order index
        int j = tid % m;      // sample index
        double x = a + (j + 1) * (b - a) / ((double)m);
        int order = i + 1;
        resultsDouble[tid] = expint_double_dev(order, x, maxIterations);
        resultsFloat[tid]  = expint_float_dev(order, (float)x, maxIterations);
    }
}

// 主机接口
void expint_cuda(int n, int m, double a, double b, int maxIterations,
                 std::vector<std::vector<float>> &gpuFloat, std::vector<std::vector<double>> &gpuDouble,
                 float &gpuTimeMs)
{
    int total = n * m;
    float *d_resultsFloat = nullptr;
    double *d_resultsDouble = nullptr;
    hipMalloc(&d_resultsFloat, total * sizeof(float));
    hipMalloc(&d_resultsDouble, total * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize = 256;
    int numBlocks = (total + blockSize - 1) / blockSize;
    expint_kernel<<<numBlocks, blockSize>>>(n, m, a, b, maxIterations, d_resultsFloat, d_resultsDouble);
    hipDeviceSynchronize();

    hipEventRecord(stop); hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTimeMs, start, stop);

    // host 结果收集
    std::vector<float> h_resultsFloat(total);
    std::vector<double> h_resultsDouble(total);
    hipMemcpy(h_resultsFloat.data(), d_resultsFloat, total * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_resultsDouble.data(), d_resultsDouble, total * sizeof(double), hipMemcpyDeviceToHost);

    // reshape 到二维
    gpuFloat.resize(n, std::vector<float>(m));
    gpuDouble.resize(n, std::vector<double>(m));
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++) {
            gpuFloat[i][j] = h_resultsFloat[i * m + j];
            gpuDouble[i][j] = h_resultsDouble[i * m + j];
        }

    hipFree(d_resultsFloat);
    hipFree(d_resultsDouble);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
